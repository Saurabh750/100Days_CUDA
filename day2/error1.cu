#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addKernel(int *c, const int *a, const int *b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
    const int arraySize = 5;
    const int a[arraySize] = {1, 2, 3, 4, 5};
    const int b[arraySize] = {10, 20, 30, 40, 50};
    int c[arraySize] = {0};

    int *d_a, *d_b, *d_c;
    
    // Manual error checking
    hipError_t err;

    err = hipMalloc((void**)&d_a, arraySize * sizeof(int));
    if (err != hipSuccess) {
        printf("CUDA malloc failed for d_a: %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMalloc((void**)&d_b, arraySize * sizeof(int));
    if (err != hipSuccess) {
        printf("CUDA malloc failed for d_b: %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMalloc((void**)&d_c, arraySize * sizeof(int));
    if (err != hipSuccess) {
        printf("CUDA malloc failed for d_c: %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(d_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("CUDA memcpy failed for d_a: %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(d_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("CUDA memcpy failed for d_b: %s\n", hipGetErrorString(err));
        return -1;
    }

    addKernel<<<1, arraySize>>>(d_c, d_a, d_b);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(c, d_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("CUDA memcpy failed for d_c: %s\n", hipGetErrorString(err));
        return -1;
    }

    printf("Result: ");
    for (int i = 0; i < arraySize; ++i)
        printf("%d ", c[i]);
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
